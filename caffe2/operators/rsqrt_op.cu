#include "hip/hip_runtime.h"
#include "caffe2/operators/rsqrt_op.h"

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T CubeCUDA(const T x) {
  return x * x * x;
}

template <typename T>
__global__ void
RSqrtGradientCUDAKernel(const int size, const T* dY, const T* Y, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, size) {
#if __CUDA_ARCH__ >= 350
    dX[i] = __ldg(dY + i) * CubeCUDA(__ldg(Y + i)) * static_cast<T>(-0.5);
#else
    dX[i] = dY[i] * CubeCUDA(Y[i]) * static_cast<T>(-0.5);
#endif
  }
}

} // namespace

template <>
template <typename T>
void RSqrtGradientFunctor<HIPContext>::Run<T>(
    const int size,
    const T* dY,
    const T* Y,
    T* dX,
    HIPContext* context) const {
  RSqrtGradientCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, dY, Y, dX);
}

REGISTER_HIP_OPERATOR(
    RSqrt,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        RSqrtFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    RSqrtGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        WithoutBroadcast<RSqrtGradientFunctor<HIPContext>>>);

} // namespace caffe2
